#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_SIZE 16

#define TIMER_CREATE(t)           \
  hipEvent_t t##_start, t##_end; \
  hipEventCreate(&t##_start);    \
  hipEventCreate(&t##_end);

#define TIMER_START(t)        \
  hipEventRecord(t##_start); \
  hipEventSynchronize(t##_start);

#define TIMER_END(t)                            \
  hipEventRecord(t##_end);                     \
  hipEventSynchronize(t##_end);                \
  hipEventElapsedTime(&t, t##_start, t##_end); \
  hipEventDestroy(t##_start);                  \
  hipEventDestroy(t##_end);

unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess)
  {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    exit(-1);
  }
#endif
  return result;
}

__global__ void sobel(unsigned char *input,
                      unsigned char *output,
                      unsigned int height,
                      unsigned int width)
{

  int x = blockIdx.x * TILE_SIZE + threadIdx.x; // column
  int y = blockIdx.y * TILE_SIZE + threadIdx.y; // row
  int Gx = 0;
  int Gy = 0;

  if (x >= 1 && x < (width - 1) && y >= 1 && y < (height - 1))
  {
    Gx = input[(y - 1) * width + x + 1] +
         2 * input[y * width + x + 1] +
         input[(y + 1) * width + x + 1] -
         (input[(y - 1) * width + x - 1] +
          2 * input[y * width + x - 1] +
          input[(y + 1) * width + x - 1]);

    Gy = input[(y - 1) * width + x - 1] +
         2 * input[(y - 1) * width + x] +
         input[(y - 1) * width + x + 1] -
         (input[(y + 1) * width + x - 1] +
          2 * input[(y + 1) * width + x] +
          input[(y + 1) * width + x + 1]);

    output[y * width + x] = (abs(Gx) + abs(Gy)) / 2;
  }
}

void gpu_function(unsigned char *input,
                  unsigned char *output,
                  unsigned int height,
                  unsigned int width)
{

  int gridXSize = 1 + ((width - 1) / TILE_SIZE);
  int gridYSize = 1 + ((height - 1) / TILE_SIZE);

  // Both are the same size (CPU/GPU).
  int size = height * width;

  // Allocate arrays in GPU memory
  checkCuda(hipMalloc((void **)&input_gpu, size * sizeof(unsigned char)));
  checkCuda(hipMalloc((void **)&output_gpu, size * sizeof(unsigned char)));
  checkCuda(hipMemset(output_gpu, 0, size * sizeof(unsigned char)));

  // Copy data to GPU
  checkCuda(hipMemcpy(input_gpu,
                       input,
                       size * sizeof(unsigned char),
                       hipMemcpyHostToDevice));

  checkCuda(hipDeviceSynchronize());

  // Execute algorithm

  dim3 dimGrid(gridXSize, gridYSize);
  dim3 dimBlock(TILE_SIZE, TILE_SIZE);

// Kernel Call
#if defined(CUDA_TIMING)
  float Ktime;
  TIMER_CREATE(Ktime);
  TIMER_START(Ktime);
#endif

  sobel<<<dimGrid, dimBlock>>>(input_gpu,
                               output_gpu,
                               height,
                               width);

  checkCuda(hipPeekAtLastError());
  checkCuda(hipDeviceSynchronize());

#if defined(CUDA_TIMING)
  TIMER_END(Ktime);
  printf("Kernel Execution Time: %f ms\n", Ktime);
#endif

  // Retrieve results from the GPU
  checkCuda(hipMemcpy(output,
                       output_gpu,
                       size * sizeof(unsigned char),
                       hipMemcpyDeviceToHost));

  // Free resources and end the program
  checkCuda(hipFree(output_gpu));
  checkCuda(hipFree(input_gpu));
}